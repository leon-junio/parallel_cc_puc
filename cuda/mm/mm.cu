
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ 
void matrixMulKernel(double* a, double* b, double* c, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width) {
        double sum = 0;
        for (int k = 0; k < width; k++) {
            double x = a[row * width + k];
            double y = b[k * width + col];
            sum += x * y;
        }
        c[row * width + col] = sum;
    }
}

void matrixMulCPU(double* a, double* b, double* c, int width) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            double sum = 0;
            for (int k = 0; k < width; k++) {
                double x = a[i * width + k];
                double y = b[k * width + j];
                sum += x * y;
            }
            c[i * width + j] = sum;
        }
    }
}

int main() {
    int width = 2000;
    size_t size = width * width * sizeof(double);

    double *h_a = (double*) malloc(size);
    double *h_b = (double*) malloc(size);
    double *h_c = (double*) malloc(size);

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            h_a[i * width + j] = i;
            h_b[i * width + j] = j;
            h_c[i * width + j] = 0;
        }
    }

    double *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y);

    matrixMulKernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Uncomment to check results
    // for (int i = 0; i < width; i++) {
    //     for (int j = 0; j < width; j++) {
    //         printf("c[%d][%d] = %f\n", i, j, h_c[i * width + j]);
    //     }
    // }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}