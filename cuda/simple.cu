
#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel function
__global__
void add(float* a, float* b){
	int index = threadIdx.x + blockIdx.x + blockDim.x;
	a[index] = a[index] + b[index];
}

int main(){
	
	int N = 2 << 10;
	float *a, *b, *a_gpu, *b_gpu;
	
	// Malloc vectors at CPU
	a = (float*) malloc(sizeof(float)*N);
	b = (float*) malloc(sizeof(float)*N);
	
	// Malloc vectors at GPU
	hipMalloc(&a_gpu, sizeof(float)*N);
	hipMalloc(&b_gpu, sizeof(float)*N);

	for(int i=0;i<N;i++){
		a[i] = 1.0f;
		b[i] = 1.0003f;
	}

	// copy vectors to GPU
	hipMemcpy(a_gpu,a,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(b_gpu,b,sizeof(float)*N,hipMemcpyHostToDevice);

	// call the kernel and choose the number of threads
	add <<< 1, N>>>(a_gpu,b_gpu);
	
	// copy results to CPU
	hipMemcpy(a,a_gpu,sizeof(float)*N,hipMemcpyDeviceToHost);

	// print the results
	for(int i=0;i<N;i++){
		printf("[%d]:%.4f", i, a[i]);
	}

	hipFree(a_gpu);
	hipFree(b_gpu);
	free(a);
	free(b);
}
